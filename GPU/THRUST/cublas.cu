#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
using namespace std;

int main(void)
{
  int N = 10000;
  thrust::host_vector<float> hA(N*N);
  thrust::host_vector<float> hB(N*N);
  thrust::host_vector<float> hC(N*N);

  thrust::default_random_engine rng;
  thrust::uniform_int_distribution<float> dist(0,128);
  
   for (int i =0; i < N*N; i++)
   {
      hA[i] = dist(rng);
      hB[i] = dist(rng);
   }
   
 
  thrust::device_vector<float> dA(N*N);
  thrust::device_vector<float> dB(N*N);
  thrust::device_vector<float> dC(N*N);
 
  dA = hA;
  dB = hB;
 
  float alpha = 1;
  float beta = 0;

  float * raw_dA, *raw_dB, *raw_dC;
  raw_dA = thrust::raw_pointer_cast(&dA[0]);
  raw_dB = thrust::raw_pointer_cast(&dB[0]);
  raw_dC = thrust::raw_pointer_cast(&dC[0]);

  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
 
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  
  hipEventRecord(start,0);
  hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N, N, N, N, &alpha, raw_dA, N, raw_dB, N, &beta, raw_dC, N);
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  float time_ms = 0.0;
  hipEventElapsedTime(&time_ms,start,stop);

  hipblasDestroy(handle);
  hC = dC;
  //thrust::copy(hC.begin(),hC.end(),ostream_iterator<float>(cout,"\n"));  
  float GFLOPS = (float)N * N * N/(1E9 * (time_ms/1E3));
  printf("Matrix Multiplication - CUBLAS 10000 x 10000\n");
  printf("GFLOPS/sec = %f \n",GFLOPS);
  printf("Time(ms): %f \t \n", time_ms);
  return 0;
}
